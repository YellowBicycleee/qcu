#include "qcd/qcu_dslash.cuh"
#include <assert.h>

BEGIN_NAMESPACE(qcu)
void DslashMV::operator()(_genvector result, _genvector src, hipStream_t stream) {

  dslash->dslashParam_->fermionOut = result;
  dslash->dslashParam_->fermionIn = src;

  dslash->preApply();
  dslash->apply();
  dslash->postApply();

  CHECK_CUDA(hipStreamSynchronize(dslash->dslashParam_->stream1));
  CHECK_CUDA(hipStreamSynchronize(dslash->dslashParam_->stream2));
}

END_NAMESPACE(qcu)