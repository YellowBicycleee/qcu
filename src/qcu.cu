#include "comm/qcu_communicator.h"
#include "mempool/qcu_mempool.h"
#include "qcd/qcu_wilson_dslash.cuh"
#include "qcu.h"
#include "qcu_macro.cuh"
#include "qcu_storage/qcu_storage.cuh"
#include <hip/hip_runtime.h>

#define PRINT_EXEC_TIME
#define PRINT_ALLOCATED_MEM_SIZE

BEGIN_NAMESPACE(qcu)
// enum DslashType { QCU_DSLASH_WILSON = 0, QCU_DSLASH_CLOVER = 1 };

class Qcu {
protected:
  bool gaugeLoaded_;

  double kappa_;
  double mass_;

  int Lx_;
  int Ly_;
  int Lz_;
  int Lt_;

  int procNx_;
  int procNy_;
  int procNz_;
  int procNt_;
  int boundaryLength_[Nd];

  void *inputGauge_;

  void *coalescedGauge_;      // coalesced gauge
  void *coalescedFermionIn_;  // coalesced fermion
  void *coalescedFermionOut_; // coalesced fermion

  void *fermionIn_;  // input fermion
  void *fermionOut_; // output fermion

  void *cloverMatrix_;
  void *cloverInvMatrix_;

  hipStream_t stream1_;
  hipStream_t stream2_;

  hipEvent_t startEvent_;
  hipEvent_t stopEvent_;

  QcuMemPool *memPool_;
  MsgHandler *msgHandler_;
  QcuComm *qcuComm_;

public:
  Qcu(int Lx, int Ly, int Lz, int Lt, int Nx, int Ny, int Nz, int Nt, double mass = 0.0)
      : Lx_(Lx), Ly_(Ly), Lz_(Lz), Lt_(Lt), procNx_(Nx), procNy_(Ny), procNz_(Nz), procNt_(Nt),
        mass_(mass), kappa_(1.0 / (2.0 * (4.0 + mass))), gaugeLoaded_(false), inputGauge_(nullptr),
        coalescedGauge_(nullptr), coalescedFermionIn_(nullptr), coalescedFermionOut_(nullptr),
        fermionIn_(nullptr), fermionOut_(nullptr), cloverMatrix_(nullptr),
        cloverInvMatrix_(nullptr), memPool_(nullptr), msgHandler_(nullptr), qcuComm_(nullptr) {
    CHECK_CUDA(hipStreamCreate(&stream1_));
    CHECK_CUDA(hipStreamCreate(&stream2_));
    CHECK_CUDA(hipEventCreate(&startEvent_));
    CHECK_CUDA(hipEventCreate(&stopEvent_));
    int vol = Lx_ * Ly_ * Lz_ * Lt_ / 2;
    CHECK_CUDA(hipMalloc(&coalescedFermionIn_, sizeof(double) * vol * 2 * Ns * Nc));
    CHECK_CUDA(hipMalloc(&coalescedFermionOut_, sizeof(double) * vol * 2 * Ns * Nc));
    msgHandler_ = new MsgHandler();
    qcuComm_ = new QcuComm(procNx_, procNy_, procNz_, procNt_);
    memPoolInit();
  }
  virtual ~Qcu() {
    CHECK_CUDA(hipStreamDestroy(stream1_));
    CHECK_CUDA(hipStreamDestroy(stream2_));
    CHECK_CUDA(hipEventDestroy(startEvent_));
    CHECK_CUDA(hipEventDestroy(stopEvent_));

    if (coalescedGauge_ != nullptr) {
      CHECK_CUDA(hipFree(coalescedGauge_));
      coalescedGauge_ = nullptr;
    }
    if (coalescedFermionIn_ != nullptr) {
      CHECK_CUDA(hipFree(coalescedFermionIn_));
      coalescedFermionIn_ = nullptr;
    }
    if (coalescedFermionOut_ != nullptr) {
      CHECK_CUDA(hipFree(coalescedFermionOut_));
      coalescedFermionOut_ = nullptr;
    }
    if (memPool_ != nullptr) {
      delete memPool_;
      memPool_ = nullptr;
    }
    if (msgHandler_ != nullptr) {
      delete msgHandler_;
      msgHandler_ = nullptr;
    }
    if (qcuComm_ != nullptr) {
      delete qcuComm_;
      qcuComm_ = nullptr;
    }
  }
  void memPoolInit() {
    memPool_ = new QcuMemPool();
    int singleVecLength = Ns * Nc;

    boundaryLength_[0] = procNx_ == 1 ? 0 : Ly_ * Lz_ * Lt_ / 2 * singleVecLength;
    boundaryLength_[1] = procNy_ == 1 ? 0 : Lx_ * Lz_ * Lt_ / 2 * singleVecLength;
    boundaryLength_[2] = procNz_ == 1 ? 0 : Lx_ * Ly_ * Lt_ / 2 * singleVecLength;
    boundaryLength_[3] = procNt_ == 1 ? 0 : Lx_ * Ly_ * Lz_ / 2 * singleVecLength;

    memPool_->allocateAllVector(boundaryLength_[0], boundaryLength_[1], boundaryLength_[2],
                                boundaryLength_[3], sizeof(double) * 2);
#ifdef PRINT_ALLOCATED_MEM_SIZE
    printf("========================\n");
    printf("Allocated memory size : \n x dim = %d\n y dim = %d\n z dim = %d\n t dim = %d\n ",
           boundaryLength_[0], boundaryLength_[1], boundaryLength_[2], boundaryLength_[3]);
    printf("========================\n");
#endif
  }

  void loadGauge(void *gauge);
  void shiftFermionStorage(void *dst, void *src, int shiftDir);
  // TODO : modify lattice size
  // void modifyLattice(int Lx, int Ly, int Lz, int Lt) {}

  // TODO : dslash wilson
  virtual void wilsonDslash(void *fermionOut, void *fermionIn, int parity);
  virtual void wilsonDslashMultiProc(void *fermionOut, void *fermionIn, int parity);
  virtual void wilsonMatMul() {}
  // TODO : dslash clover
  virtual void cloverDslash() {}
  virtual void cloverMatMul() {}
  // TODO : invert (cg inverter)
  virtual void qcuInvert() {}
};

void Qcu::wilsonDslash(void *fermionOut, void *fermionIn, int parity) {
  int daggerFlag = 0;

  fermionIn_ = fermionIn;
  fermionOut_ = fermionOut;
  shiftFermionStorage(coalescedFermionIn_, fermionIn_, TO_COALESCE);

  DslashParam dslashParam(coalescedFermionIn_, coalescedFermionOut_, coalescedGauge_, Lx_, Ly_, Lz_,
                          Lt_, parity, procNx_, procNy_, procNz_, procNt_, daggerFlag, memPool_, msgHandler_,
                          qcuComm_);

  CHECK_CUDA(hipEventRecord(startEvent_, stream1_));
  WilsonDslash dslash(&dslashParam, 256, stream1_);
  dslash.apply();
  CHECK_CUDA(hipEventRecord(stopEvent_, stream1_));
  CHECK_CUDA(hipEventSynchronize(stopEvent_));

#ifdef PRINT_EXEC_TIME
  float elapsedTime;
  CHECK_CUDA(hipEventElapsedTime(&elapsedTime, startEvent_, stopEvent_));
  printf("Recorded time : %f s\n", elapsedTime / 1000);
#endif

  shiftFermionStorage(fermionOut_, coalescedFermionOut_, TO_NON_COALESCE);
}

void Qcu::wilsonDslashMultiProc(void *fermionOut, void *fermionIn, int parity) {
  // if (procNx_ == 1 && procNy_ == 1 && procNz_ == 1 && procNt_ == 1) {
  //   return;
  // }
  // shiftStorage
  int daggerFlag = 0;
  fermionIn_ = fermionIn;
  fermionOut_ = fermionOut;
  shiftFermionStorage(coalescedFermionIn_, fermionIn_, TO_COALESCE);
  DslashParam dslashParam(coalescedFermionIn_, coalescedFermionOut_, coalescedGauge_, Lx_, Ly_, Lz_,
                          Lt_, parity, procNx_, procNy_, procNz_, procNt_, daggerFlag, memPool_, msgHandler_,
                          qcuComm_);
  CHECK_CUDA(hipEventRecord(startEvent_, stream1_));
  WilsonDslash dslash(&dslashParam, 256, stream1_, stream2_);

  dslash.preApply();
  dslash.apply();
  dslash.postApply();

  CHECK_CUDA(hipEventRecord(stopEvent_, stream1_));
  CHECK_CUDA(hipEventSynchronize(stopEvent_));

#ifdef PRINT_EXEC_TIME
  float elapsedTime;
  CHECK_CUDA(hipEventElapsedTime(&elapsedTime, startEvent_, stopEvent_));
  printf("Recorded time : %f s\n", elapsedTime / 1000);
#endif
  // shiftStorage
  shiftFermionStorage(fermionOut_, coalescedFermionOut_, TO_NON_COALESCE);
}

void Qcu::loadGauge(void *gauge) {
  if (!gaugeLoaded_ && coalescedGauge_ == nullptr) {
    CHECK_CUDA(hipMalloc(&coalescedGauge_,
                          sizeof(double) * Nd * Lx_ * Ly_ * Lz_ * Lt_ * (Nc - 1) * Nc * 2));
  }

  shiftGaugeStorageTwoDouble(coalescedGauge_, gauge, TO_COALESCE, Lx_, Ly_, Lz_, Lt_);
  gaugeLoaded_ = true;
}

// TODO : 消除不必要代码
void Qcu::shiftFermionStorage(void *dst, void *src, int shiftDir) {
  if (shiftDir == TO_COALESCE) {
    shiftVectorStorageTwoDouble(dst, src, TO_COALESCE, Lx_, Ly_, Lz_, Lt_);
  } else if (shiftDir == TO_NON_COALESCE) {
    shiftVectorStorageTwoDouble(dst, src, TO_NON_COALESCE, Lx_, Ly_, Lz_, Lt_);
  }
}
END_NAMESPACE(qcu)

static qcu::Qcu *qcu_ptr = nullptr;

void initGridSize(QcuGrid_t *grid, QcuParam *p_param, void *gauge, void *fermion_in,
                  void *fermion_out) {
  if (qcu_ptr == nullptr) {
    qcu_ptr = new qcu::Qcu(p_param->lattice_size[0], p_param->lattice_size[1],
                           p_param->lattice_size[2], p_param->lattice_size[3], grid->grid_size[0],
                           grid->grid_size[1], grid->grid_size[2], grid->grid_size[3]);
  }
}

void destroyQcu() {
  if (qcu_ptr != nullptr) {
    delete qcu_ptr;
    qcu_ptr = nullptr;
  }
}

void dslashQcu(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param, int parity) {
  qcu_ptr->loadGauge(gauge);
  // qcu_ptr->wilsonDslash(fermion_out, fermion_in, parity);
  qcu_ptr->wilsonDslashMultiProc(fermion_out, fermion_in, parity);
}

void fullDslashQcu(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param,
                   int dagger_flag) {}
void cg_inverter(void *x_vector, void *b_vector, void *gauge, QcuParam *param, double p_max_prec,
                 double p_kappa) {}

// TODO : delete parameter param.
void loadQcuGauge(void *gauge, QcuParam *param) {}